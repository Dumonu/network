#include "hip/hip_runtime.h"
extern "C" {
#include "cudaOps.h"
#include "node.h"
}

__global__ //kernel
void cudaCalcLayerValues(struct Node** nodes, int size)
{
    int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;

    while(threadIndex < size)
    {
        calcNodeValue(nodes[threadIndex]);
        threadIndex += blockDim.x * gridDim.x;
    }
    
}

extern "C"
void cudaCallCalcLayerKernel(const uint blockCount, const uint threadsPerBlock,
        struct Node** nodes, int size)
{
    cudaCalcLayerValues<<<blockCount, threadsPerBlock>>>(nodes, size);
}
